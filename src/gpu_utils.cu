#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
#include <stddef.h>
#include <stdbool.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hipsolver.h>


int nDevices = -1;

hipblasHandle_t handle_cublas;

hipsolverHandle_t handle_cusolver;


typedef size_t devptr_t;
  // device pointer arrays

devptr_t *dev_ptrs;

// Fortran interface macro funx_
#define F90(name) name##_

#if CUDART_VERSION >= 12000
static const char *
cublasGetErrorString(hipblasStatus_t err) {
  return cublasGetStatusString(err);
}
#else
static const char *
cublasGetErrorString(hipblasStatus_t err) {
  switch (err) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    default:
      return "UNKNOWN_ERROR";
  }
}
#endif  // CUDART_VERSION >= 12000

static const char *cusolverGetErrorString(hipsolverStatus_t status) {
  switch (status) {
    case HIPSOLVER_STATUS_SUCCESS:
      return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
      return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
      return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
      return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
      return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
      return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
      return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
      return "HIPSOLVER_STATUS_MAPPING_ERROR";
    default:
      return "UNKNOWN_ERROR";
  }
}

inline void hipError_t(hipError_t err, const char *file, const int line) {
  if (err != hipSuccess) {
    printf("*** CUDA Error in %s at line %d : %s -code: %i \n", file, line,
           hipGetErrorString(err), err);
    hipDeviceReset();
    exit(-1);
  }
}

inline void checkCudaError(const char *file, const int line)
{
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("*** CUDA Error in %s, line %i\n", file, line);
    printf("%s\n", hipGetErrorString(error));
    hipDeviceReset();
    exit(-1);
  }
}

inline void cublasError(hipblasStatus_t err, const char *file, const int line) {
  if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("*** CUBLAS Error in %s at line %d : %s -code: %i\n", file, line,
           cublasGetErrorString(err), err);
    hipDeviceReset();
    exit(-1);
  }
}

inline void cusolverError(hipsolverStatus_t err, const char *file,
                          const int line) {
  if (err != HIPSOLVER_STATUS_SUCCESS) {
    printf("*** CUSOLVER Error in %s at line %d : %s -code: %i \n", file, line,
           cusolverGetErrorString(err), err);
    exit(-1);
  }
}


#define CHECK_FOR_ERROR() (checkCudaError(__FILE__, __LINE__))
#define HANDLE_CUDA( err ) (hipError_t( err, __FILE__, __LINE__ ))
#define HANDLE_CUBLAS( err ) (cublasError( err, __FILE__, __LINE__ ))
#define HANDLE_CUSOLVER( err ) (cusolverError( err, __FILE__, __LINE__ ))

// conj complex
__global__ void cmplxConj_gpu( hipDoubleComplex *vec1, hipDoubleComplex *vec2, const int n )
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  while( tid < n){
    vec2[tid] = hipConj(vec1[tid]);
    tid += stride;
  }
}
// init cuda and cublas cusolver
//extern "C" void F90(cudalib_init)(int *dev_id) {
extern "C" void F90(cudalib_init)(void) {
  //HANDLE_CUDA(hipSetDevice(*dev_id));
  //HANDLE_CUDA(hipDeviceReset());
  HANDLE_CUBLAS(hipblasCreate(&handle_cublas));
  HANDLE_CUSOLVER(hipsolverDnCreate(&handle_cusolver));
}

extern "C" void F90(cudalib_free)(void) {
  if (handle_cublas) hipblasDestroy(handle_cublas);
  if (handle_cusolver) hipsolverDnDestroy(handle_cusolver);
}

extern "C" void F90(set_gpu)(int *dev_id) {
  HANDLE_CUDA(hipSetDevice(*dev_id));
}

extern "C" void F90(alloc_cublas)(const int *n, const int *elemSize,
                                  devptr_t *devicePtr) {
  void *ptr;
  HANDLE_CUDA(hipMalloc((void **)&ptr, (*n) * (*elemSize)));
  *devicePtr = (devptr_t)(ptr);
}

extern "C" void F90(free_cublas)(devptr_t *devicePtr) {
  HANDLE_CUDA(hipFree(devicePtr));
}

extern "C" void F90(getgpusavailable)(int *result){
  HANDLE_CUDA(hipGetDeviceCount(result));
  nDevices = *result;
}

extern "C" void F90(malloc_gpu)(long *ptr, int *length, int *elemSize) {
  double *temp;
  long l = *length;
  long e = *elemSize;
  long s = l * e;
  HANDLE_CUDA(hipMalloc((void **)&temp, s));
  *ptr = (long)(temp);
}

extern "C" void F90(memcpy_gpu)(double *host_ptr, long *device_ptr,
                                 int *length, int *elemSize, int *flag) {
  long l = *length;
  long e = *elemSize;
  long size = l * e;
  double *ptr_d = (double *)(*device_ptr);
  if (*flag == 1) {
    // copy from CPU to GPU
    HANDLE_CUDA(hipMemcpy(ptr_d, host_ptr, size, hipMemcpyHostToDevice));
  } else if (*flag == 2) {
    // copy from GPU to CPU
    HANDLE_CUDA(hipMemcpy(host_ptr, ptr_d, size, hipMemcpyDeviceToHost));
  } else {
    printf(" Wrong parameter.... \n  Copy flag : %d \n ", *flag);
    exit(-1);
  }
}

extern "C" void F90(free_gpu)(long *ptr) {
  double *temp;
  temp = (double *)(*ptr);
  CHECK(hipFree(temp) == hipSuccess);
}


extern "C" void F90(zhegvd_gpu)(int *type, char *job, char *uplo_h, int *len,
                                long *dev_A, long *dev_B, long *dev_eigs,
                                int *info) {
  hipDoubleComplex *A = (hipDoubleComplex *)(*dev_A);
  hipDoubleComplex *B = (hipDoubleComplex *)(*dev_B);
  double *eigs = (double *)(*dev_eigs);

  hipDoubleComplex *d_work = NULL;
  int *devInfo = NULL;

  hipsolverEigType_t itype;
  hipsolverEigMode_t jobz;
  hipblasFillMode_t uplo;

  char zjob = *job;
  char uplo_c = *uplo_h;
  int n = *len;
  int lda = n;
  int ldb = n;

  if (zjob == 'v' || zjob == 'V')
    jobz = HIPSOLVER_EIG_MODE_VECTOR;
  else
    jobz = HIPSOLVER_EIG_MODE_NOVECTOR;

  if (uplo_c == 'U' || uplo_c == 'u')
    uplo = HIPBLAS_FILL_MODE_UPPER;
  else
    uplo = HIPBLAS_FILL_MODE_LOWER;

  if (*type == 1) itype = HIPSOLVER_EIG_TYPE_1;
  if (*type == 2) itype = HIPSOLVER_EIG_TYPE_2;
  if (*type == 3) itype = HIPSOLVER_EIG_TYPE_3;

  int lwork = -1;
  HANDLE_CUSOLVER(hipsolverDnZhegvd_bufferSize(handle_cusolver,
                                              itype,
                                              jobz,
                                              uplo,
                                              n,
                                              A,
                                              lda,
                                              B,
                                              ldb,
                                              eigs,
                                              &lwork));

  HANDLE_CUDA(hipMalloc((void **)&d_work, sizeof(hipDoubleComplex) * lwork));

  HANDLE_CUDA(hipMalloc((void **)&devInfo, sizeof(int)));

  HANDLE_CUSOLVER(hipsolverDnZhegvd(handle_cusolver,
                                   itype,
                                   jobz,
                                   uplo,
                                   n,
                                   A,
                                   lda,
                                   B,
                                   ldb,
                                   eigs,
                                   d_work,
                                   lwork,
                                   devInfo));

  HANDLE_CUDA(hipMemcpy(info, devInfo, sizeof(int), hipMemcpyDeviceToHost));

  HANDLE_CUDA(hipFree(d_work));
  HANDLE_CUDA(hipFree(devInfo));
}
